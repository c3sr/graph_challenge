#include "hip/hip_runtime.h"
/*! Display system topology

*/

#include <fmt/format.h>
#include <iostream>

#include "clara/clara.hpp"
#include "pangolin/pangolin.cuh"
#include "pangolin/pangolin.hpp"

int main(int argc, char **argv) {

  pangolin::init();

  bool help = false;
  bool debug = false;
  bool verbose = false;

  clara::Parser cli;
  cli = cli | clara::Help(help);
  cli = cli | clara::Opt(debug)["--debug"]("print debug messages to stderr");
  cli = cli |
        clara::Opt(verbose)["--verbose"]("print verbose messages to stderr");

  auto result = cli.parse(clara::Args(argc, argv));
  if (!result) {
    LOG(error, "Error in command line: {}", result.errorMessage());
    exit(1);
  }

  if (help) {
    std::cout << cli;
    return 0;
  }

  // set logging level
  if (verbose) {
    pangolin::logger::set_level(pangolin::logger::Level::TRACE);
  } else if (debug) {
    pangolin::logger::set_level(pangolin::logger::Level::DEBUG);
  }

  // log command line before much else happens
  {
    std::string cmd;
    for (int i = 0; i < argc; ++i) {
      if (i != 0) {
        cmd += " ";
      }
      cmd += argv[i];
    }
    LOG(debug, cmd);
  }
  LOG(debug, "pangolin version: {}.{}.{}", PANGOLIN_VERSION_MAJOR,
      PANGOLIN_VERSION_MINOR, PANGOLIN_VERSION_PATCH);
  LOG(debug, "pangolin branch:  {}", PANGOLIN_GIT_REFSPEC);
  LOG(debug, "pangolin sha:     {}", PANGOLIN_GIT_HASH);
  LOG(debug, "pangolin changes: {}", PANGOLIN_GIT_LOCAL_CHANGES);

#ifndef NDEBUG
  LOG(warn, "Not a release build");
#endif

  auto cpus = pangolin::topology::get_cpus();

  for (const auto cpu : cpus) {
    fmt::print(stdout, "cpu {}\n", cpu);
    auto numas = pangolin::topology::cpu_numa_affinity({cpu});
    for (const auto numa : numas) {
      fmt::print(stdout, "\tnuma {}\n", numa);
    }
  }

  return 0;
}
