#include "hip/hip_runtime.h"
/*!

Count triangles using the per-edge binary search

*/

#include <iostream>
#include <thread>
#include <vector>

#include <nvToolsExt.h>

#include <clara/clara.hpp>
#include <fmt/format.h>

#include "pangolin/algorithm/tc_edge_binary.cuh"
#include "pangolin/bounded_buffer.hpp"
#include "pangolin/configure.hpp"
#include "pangolin/cuda_cxx/stream.hpp"
#include "pangolin/file/edge_list_file.hpp"
#include "pangolin/init.hpp"
#include "pangolin/sparse/csr_coo.hpp"

// Buffer is a BoundedBuffer with two entries (double buffer)
template <typename T> using Buffer = pangolin::BoundedBuffer<T, 2>;
using pangolin::Stream;

template <typename V> void print_vec(const V &vec, const std::string &sep) {
  for (const auto &e : vec) {
    fmt::print("{}{}", sep, e);
  }
}

template <typename Edge> void produce(const std::string path, Buffer<std::vector<Edge>> &queue) {
  double readTime = 0, queueTime = 0;
  pangolin::EdgeListFile file(path);

  std::vector<Edge> edges;

  while (true) {
    auto readStart = std::chrono::system_clock::now();
    size_t readCount = file.get_edges(edges, 500);
    auto readEnd = std::chrono::system_clock::now();
    readTime += (readEnd - readStart).count() / 1e9;
    SPDLOG_TRACE(pangolin::logger::console(), "reader: read {} edges", edges.size());
    if (0 == readCount) {
      break;
    }

    auto queueStart = std::chrono::system_clock::now();
    queue.push(std::move(edges));
    auto queueEnd = std::chrono::system_clock::now();
    queueTime += (queueEnd - queueStart).count() / 1e9;
    SPDLOG_TRACE(pangolin::logger::console(), "reader: pushed edges");
  }

  SPDLOG_TRACE(pangolin::logger::console(), "reader: closing queue");
  queue.close();
  LOG(debug, "reader: {}s I/O, {}s blocked", readTime, queueTime);
}

template <typename Mat> void consume(Buffer<std::vector<typename Mat::edge_type>> &queue, Mat &mat) {
  typedef typename Mat::index_type Index;
  typedef typename Mat::edge_type Edge;

  double queueTime = 0, csrTime = 0;
  auto upperTriangular = [](const Edge &e) { return e.src < e.dst; };

  // keep grabbing while queue is filling
  Index maxNode = 0;
  while (true) {
    std::vector<Edge> edges;
    bool popped;
    SPDLOG_TRACE(pangolin::logger::console(), "builder: trying to pop...");
    auto queueStart = std::chrono::system_clock::now();
    edges = queue.pop(popped);
    auto queueEnd = std::chrono::system_clock::now();
    queueTime += (queueEnd - queueStart).count() / 1e9;
    if (popped) {
      SPDLOG_TRACE(pangolin::logger::console(), "builder: popped {} edges", edges.size());
      auto csrStart = std::chrono::system_clock::now();
      for (const auto &edge : edges) {
        maxNode = max(edge.src, maxNode);
        maxNode = max(edge.dst, maxNode);
        if (upperTriangular(edge)) {
          // SPDLOG_TRACE(pangolin::logger::console(), "{} {}", edge.src, edge.dst);
          mat.add_next_edge(edge);
        }
      }
      auto csrEnd = std::chrono::system_clock::now();
      csrTime += (csrEnd - csrStart).count() / 1e9;
    } else {
      SPDLOG_TRACE(pangolin::logger::console(), "builder: no edges after pop");
      assert(queue.empty());
      assert(queue.closed());
      break;
    }
  }

  auto csrStart = std::chrono::system_clock::now();
  mat.finish_edges(maxNode);
  auto csrEnd = std::chrono::system_clock::now();
  csrTime += (csrEnd - csrStart).count() / 1e9;

  LOG(debug, "builder: {}s csr {}s blocked", csrTime, queueTime);
}

struct RunOptions {
  int iters;
  std::vector<int> gpus;
  std::string path; //!< path for graph
  std::string sep;  //!< seperator for output
  int blockSize;
  int coarsening;

  bool readMostly;
  bool accessedBy;
  bool prefetchAsync;
  bool shrinkToFit;
  bool preCountBarrier;
};

void print_header(const RunOptions &opts) {
  fmt::print("bmark{0}bs{0}gpus{0}graph{0}nodes{0}edges{0}tris", opts.sep);
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}total_time{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}total_teps{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}gpu_time{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}gpu_teps{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}count_time{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}count_teps{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}kernel_time{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}kernel_teps{}", opts.sep, i);
  }
  fmt::print("\n");
}

template <typename Index> int run(RunOptions &opts) {
  typedef pangolin::DiEdge<Index> Edge;

  auto gpus = opts.gpus;
  if (gpus.empty()) {
    LOG(warn, "no GPUs provided on command line, using GPU 0");
    gpus.push_back(0);
  }

  // create a stream for each GPU
  std::vector<Stream> streams;
  for (const auto &gpu : gpus) {
    streams.push_back(Stream(gpu));
    LOG(debug, "created stream {} for gpu {}", streams.back(), gpu);
  }

  std::vector<double> totalTimes;
  std::vector<double> gpuTimes;
  std::vector<double> countTimes;
  std::vector<double> kernelTimes;
  uint64_t nnz;
  uint64_t numRows;
  uint64_t tris;
  // create csr and count `opts.iters` times
  for (int i = 0; i < opts.iters; ++i) {

    // read data
    const auto totalStart = std::chrono::system_clock::now();
    Buffer<std::vector<Edge>> queue;
    pangolin::CSRCOO<Index> csr;
    // start a thread to read the matrix data
    LOG(debug, "start disk reader");
    std::thread reader(produce<Edge>, opts.path, std::ref(queue));
    // start a thread to build the matrix
    LOG(debug, "start csr build");
    std::thread builder(consume<pangolin::CSRCOO<Index>>, std::ref(queue), std::ref(csr));
    // consume(queue, csr, &readerActive);
    LOG(debug, "waiting for disk reader...");
    reader.join();
    LOG(debug, "waiting for CSR builder...");
    builder.join();
    assert(queue.empty());

    if (opts.shrinkToFit) {
      LOG(debug, "shrink CSR");
      csr.shrink_to_fit();
    }

    double elapsed = (std::chrono::system_clock::now() - totalStart).count() / 1e9;
    LOG(info, "io/csr time {}s", elapsed);
    LOG(debug, "CSR nnz = {} rows = {}", csr.nnz(), csr.num_rows());
    LOG(debug, "CSR cap = {}MB size = {}MB", csr.capacity_bytes() / 1024 / 1024, csr.size_bytes() / 1024 / 1024);

    const auto gpuStart = std::chrono::system_clock::now();

    // read-mostly
    nvtxRangePush("read-mostly");
    auto start = std::chrono::system_clock::now();
    if (opts.readMostly) {
      csr.read_mostly();
    }
    elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    nvtxRangePop();
    LOG(info, "read-mostly CSR time {}s", elapsed);

    // accessed-by
    start = std::chrono::system_clock::now();
    if (opts.accessedBy) {
      for (const auto &gpu : gpus) {
        csr.accessed_by(gpu);
      }
    }
    elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    LOG(info, "accessed-by CSR time {}s", elapsed);

    // prefetch
    start = std::chrono::system_clock::now();
    if (opts.prefetchAsync) {
      for (size_t gpuIdx = 0; gpuIdx < gpus.size(); ++gpuIdx) {
        auto &gpu = gpus[gpuIdx];
        hipStream_t stream = streams[gpuIdx].stream();
        csr.prefetch_async(gpu, stream);
      }
    }
    elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    LOG(info, "prefetch CSR time {}s", elapsed);

    if (opts.preCountBarrier) {
      LOG(debug, "sync streams after hints");
      for (auto &stream : streams) {
        stream.sync();
      }
    }

    // count triangles
    nvtxRangePush("count");
    const auto countStart = std::chrono::system_clock::now();

    // create async counters
    std::vector<pangolin::BinaryTC> counters;
    for (size_t gpuIdx = 0; gpuIdx < gpus.size(); ++gpuIdx) {
      auto dev = gpus[gpuIdx];
      hipStream_t stream = streams[gpuIdx].stream();
      LOG(debug, "create device {} counter", dev);
      counters.push_back(std::move(pangolin::BinaryTC(dev, stream)));
    }

    // determine the number of edges per gpu
    const size_t edgesPerGPU = (csr.nnz() + gpus.size() - 1) / gpus.size();
    LOG(debug, "{} edges per GPU", edgesPerGPU);

    // launch counting operations
    size_t edgeStart = 0;
    for (auto &counter : counters) {
      const size_t edgeStop = std::min(edgeStart + edgesPerGPU, csr.nnz());
      const size_t numEdges = edgeStop - edgeStart;
      LOG(debug, "start async count on GPU {} ({} edges)", counter.device(), numEdges);
      counter.count_async(csr.view(), numEdges, edgeStart, opts.blockSize, opts.coarsening);
      edgeStart += edgesPerGPU;
    }

    // wait for counting operations to finish
    uint64_t total = 0;
    for (auto &counter : counters) {
      LOG(debug, "wait for counter on GPU {}", counter.device());
      counter.sync();
      total += counter.count();
    }
    const auto stop = std::chrono::system_clock::now();
    nvtxRangePop(); // count
    LOG(info, "{} triangles", total);

    // record graph stats
    tris = total;
    nnz = csr.nnz();
    numRows = csr.num_rows();

    const double totalElapsed = (stop - totalStart).count() / 1e9;
    const double gpuElapsed = (stop - gpuStart).count() / 1e9;
    const double countElapsed = (stop - countStart).count() / 1e9;
    LOG(info, "total time {}s ({} teps)", totalElapsed, nnz / totalElapsed);
    LOG(info, "gpu time   {}s ({} teps)", gpuElapsed, nnz / gpuElapsed);
    LOG(info, "count time {}s ({} teps)", countElapsed, nnz / countElapsed);
    totalTimes.push_back(totalElapsed);
    gpuTimes.push_back(gpuElapsed);
    countTimes.push_back(countElapsed);

    for (auto &counter : counters) {
      double secs = counter.kernel_time();
      int dev = counter.device();
      LOG(info, "gpu {} kernel time {}s ({} teps)", dev, secs, nnz / secs);
    }
    if (counters.size() == 1) {
      kernelTimes.push_back(counters[0].kernel_time());
    } else {
      kernelTimes.push_back(0);
    }
  }

  if (opts.iters > 0) {
    fmt::print("binary");
    fmt::print("{}{}", opts.sep, opts.blockSize);
    std::string gpuStr;
    for (auto gpu : gpus) {
      gpuStr += std::to_string(gpu);
    }
    fmt::print("{}{}", opts.sep, gpuStr);
    fmt::print("{}{}", opts.sep, opts.path);
    fmt::print("{}{}", opts.sep, numRows);
    fmt::print("{}{}", opts.sep, nnz);
    fmt::print("{}{}", opts.sep, tris);

    print_vec(totalTimes, opts.sep);
    for (const auto &s : totalTimes) {
      fmt::print("{}{}", opts.sep, nnz / s);
    }
    print_vec(gpuTimes, opts.sep);
    for (const auto &s : gpuTimes) {
      fmt::print("{}{}", opts.sep, nnz / s);
    }
    print_vec(countTimes, opts.sep);
    for (const auto &s : countTimes) {
      fmt::print("{}{}", opts.sep, nnz / s);
    }
    print_vec(kernelTimes, opts.sep);
    for (const auto &s : kernelTimes) {
      fmt::print("{}{}", opts.sep, nnz / s);
    }

    fmt::print("\n");
  }

  return 0;
}

int main(int argc, char **argv) {

  pangolin::init();

  RunOptions opts;
  opts.sep = ",";
  opts.blockSize = 512;
  opts.coarsening = 1;
  opts.iters = 1;
  opts.shrinkToFit = false;
  opts.readMostly = false;
  opts.accessedBy = false;
  opts.prefetchAsync = false;
  opts.preCountBarrier = true;

  bool help = false;
  bool debug = false;
  bool verbose = false;
  bool onlyPrintHeader = false;
  bool wide = false;

  clara::Parser cli;
  cli = cli | clara::Help(help);
  cli = cli | clara::Opt(debug)["--debug"]("print debug messages to stderr");
  cli = cli | clara::Opt(verbose)["--verbose"]("print verbose messages to stderr");
  cli = cli | clara::Opt(onlyPrintHeader)["--header"]("print the header for the times output and quit");
  cli = cli | clara::Opt(wide)["--wide"]("64-bit node IDs");
  cli = cli | clara::Opt(opts.gpus, "dev ids")["-g"]("gpus to use");
  cli = cli | clara::Opt(opts.coarsening, "coarsening")["-c"]("Number of elements per thread");
  cli = cli | clara::Opt(opts.blockSize, "block-dim")["--bs"]("Number of threads in a block");
  cli = cli | clara::Opt(opts.shrinkToFit)["--shrink-to-fit"]("shrink allocations to fit data");
  cli = cli | clara::Opt(opts.readMostly)["--read-mostly"]("mark data as read-mostly by all gpus before kernel");
  cli = cli | clara::Opt(opts.accessedBy)["--accessed-by"]("mark data as accessed-by all GPUs before kernel");
  cli = cli | clara::Opt(opts.prefetchAsync)["--prefetch-async"]("prefetch data to all GPUs before kernel");
  cli = cli | clara::Opt(opts.iters, "N")["-n"]("number of counts");
  cli = cli | clara::Arg(opts.path, "graph file")("Path to adjacency list").required();

  auto result = cli.parse(clara::Args(argc, argv));
  if (!result) {
    LOG(error, "Error in command line: {}", result.errorMessage());
    exit(1);
  }

  if (help) {
    std::cout << cli;
    return 0;
  }

  // set logging level
  if (verbose) {
    pangolin::logger::set_level(pangolin::logger::Level::TRACE);
  } else if (debug) {
    pangolin::logger::set_level(pangolin::logger::Level::DEBUG);
  }

  // log command line before much else happens
  {
    std::string cmd;
    for (int i = 0; i < argc; ++i) {
      if (i != 0) {
        cmd += " ";
      }
      cmd += argv[i];
    }
    LOG(debug, cmd);
  }
  LOG(debug, "pangolin version: {}.{}.{}", PANGOLIN_VERSION_MAJOR, PANGOLIN_VERSION_MINOR, PANGOLIN_VERSION_PATCH);
  LOG(debug, "pangolin branch:  {}", PANGOLIN_GIT_REFSPEC);
  LOG(debug, "pangolin sha:     {}", PANGOLIN_GIT_HASH);
  LOG(debug, "pangolin changes: {}", PANGOLIN_GIT_LOCAL_CHANGES);

#ifndef NDEBUG
  LOG(warn, "Not a release build");
#endif

  if (onlyPrintHeader) {
    print_header(opts);
    return 0;
  }
  if (wide) {
    return run<uint64_t>(opts);
  } else {
    return run<uint32_t>(opts);
  }
}
