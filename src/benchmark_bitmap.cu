#include "hip/hip_runtime.h"
/*!

Count triangles using the per-edge binary search

*/

#include <iostream>

#include <nvToolsExt.h>

#include <clara/clara.hpp>
#include <fmt/format.h>

#include "pangolin/algorithm/tc_vertex_bitvector.cuh"
#include "pangolin/configure.hpp"
#include "pangolin/file/edge_list_file.hpp"
#include "pangolin/init.hpp"
#include "pangolin/sparse/csr.hpp"

struct RunOptions {
  std::string path; //!< path for graph
  std::string sep;  //!< seperator for output
  std::vector<int> gpus;
  int dimBlock;
  int iters;

  bool readMostly;
  bool accessedBy;
  bool prefetchAsync;
};

template <typename Index> int run(RunOptions &opts) {

  typedef typename pangolin::DiEdge<Index> Edge;

  auto gpus = opts.gpus;
  if (gpus.empty()) {
    LOG(warn, "no GPUs provided on command line, using GPU 0");
    gpus.push_back(0);
  }

  // read data
  auto start = std::chrono::system_clock::now();
  pangolin::EdgeListFile file(opts.path);

  std::vector<Edge> edges;
  std::vector<Edge> fileEdges;
  while (file.get_edges(fileEdges, 500)) {
    edges.insert(edges.end(), fileEdges.begin(), fileEdges.end());
  }
  double elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
  LOG(info, "read_data time {}s", elapsed);
  LOG(debug, "read {} edges", edges.size());

  // record various times for each iteration
  std::vector<double> iterationTimes(opts.iters, 0);   // the total wall time elapsed during each iteration
  std::vector<double> competitionTimes(opts.iters, 0); // the actual competition time
  std::vector<double> csrTimes(opts.iters, 0);         // time taken to build the CSR
  std::vector<double> kernelTimes(opts.iters,
                                  0); // just the triangle counting kernel
  std::vector<double> countTimes(
      opts.iters, 0); // the wall time elapsed configuring, launching, and waiting for counting operations
  std::vector<double> readMostlyTimes(opts.iters,
                                      0);           // times elapsed during read-mostly
  std::vector<double> prefetchTimes(opts.iters, 0); // times elapsed during prefetch
  std::vector<double> accessedByTimes(opts.iters,
                                      0); // times elapsed during accessed-by
  std::vector<double> counterCtorTimes(opts.iters,
                                       0); // time taken to construct counters

  uint64_t nnz;
  uint64_t tris;
  // create csr and count `opts.iters` times
  for (int i = 0; i < opts.iters; ++i) {
    auto iterStart = std::chrono::system_clock::now();
    auto competitionStart = std::chrono::system_clock::now();
    // create csr
    auto upperTriangularFilter = [](Edge e) { return e.src < e.dst; };
    auto lowerTriangularFilter = [](Edge e) { return e.src > e.dst; };
    auto csr = pangolin::CSR<uint64_t>::from_edges(edges.begin(), edges.end(), upperTriangularFilter);
    LOG(debug, "CSR nnz = {} rows = {}", csr.nnz(), csr.num_rows());
    elapsed = (std::chrono::system_clock::now() - iterStart).count() / 1e9;
    LOG(info, "create CSR time {}s", elapsed);
    csrTimes[i] = elapsed;

    // read-mostly
    nvtxRangePush("read-mostly");
    start = std::chrono::system_clock::now();
    if (opts.readMostly) {
      csr.read_mostly();
      for (const auto &gpu : gpus) {
        CUDA_RUNTIME(hipSetDevice(gpu));
        CUDA_RUNTIME(hipDeviceSynchronize());
      }
    }
    elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    nvtxRangePop();
    LOG(info, "read-mostly CSR time {}s", elapsed);
    readMostlyTimes[i] = elapsed;

    // accessed-by
    start = std::chrono::system_clock::now();
    if (opts.accessedBy) {
      for (const auto &gpu : gpus) {
        csr.accessed_by(gpu);
        CUDA_RUNTIME(hipSetDevice(gpu));
        CUDA_RUNTIME(hipDeviceSynchronize());
      }
    }
    elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    LOG(info, "accessed-by CSR time {}s", elapsed);
    accessedByTimes[i] = elapsed;

    // prefetch
    start = std::chrono::system_clock::now();
    if (opts.prefetchAsync) {
      for (const auto &gpu : gpus) {
        csr.prefetch_async(gpu);
        CUDA_RUNTIME(hipSetDevice(gpu));
        CUDA_RUNTIME(hipDeviceSynchronize());
      }
    }
    elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    LOG(info, "prefetch CSR time {}s", elapsed);
    prefetchTimes[i] = elapsed;

    // create async counters
    const auto counterCtorStart = std::chrono::system_clock::now();
    std::vector<pangolin::VertexBitvectorTC> counters;
    for (int dev : gpus) {
      LOG(debug, "create device {} counter", dev);
      counters.push_back(std::move(pangolin::VertexBitvectorTC(dev)));
    }
    elapsed = (std::chrono::system_clock::now() - counterCtorStart).count() / 1e9;
    counterCtorTimes[i] = elapsed;

    const auto countStart = std::chrono::system_clock::now();

    // determine the number of rows per gpu
    const size_t rowsPerGPU = (csr.num_rows() + gpus.size() - 1) / gpus.size();
    LOG(debug, "{} rows per GPU", rowsPerGPU);

    // launch counting operations
    size_t rowStart = 0;
    for (auto &counter : counters) {
      const size_t rowStop = std::min(rowStart + rowsPerGPU, csr.nnz());
      const size_t numRows = rowStop - rowStart;
      LOG(debug, "start async count on GPU {} ({} rows)", counter.device(), numRows);
      counter.count_async(csr.view(), rowStart, numRows, opts.dimBlock);
      rowStart += rowsPerGPU;
    }

    // wait for counting operations to finish
    uint64_t total = 0;
    for (auto &counter : counters) {
      LOG(debug, "wait for counter on GPU {}", counter.device());
      counter.sync();
      total += counter.count();
    }

    const auto countStop = std::chrono::system_clock::now();

    elapsed = (countStop - countStart).count() / 1e9;
    LOG(info, "count time {}s", elapsed);
    LOG(info, "{} triangles ({} teps)", total, csr.nnz() / elapsed);
    countTimes[i] = elapsed;
    elapsed = (countStop - competitionStart).count() / 1e9;
    competitionTimes[i] = elapsed;
    elapsed = (countStop - iterStart).count() / 1e9;
    iterationTimes[i] = elapsed;

    tris = total;
    nnz = csr.nnz();
  }

  std::cout << opts.path << opts.sep << nnz << opts.sep << tris << opts.sep << opts.dimBlock;
  for (auto t : countTimes) {
    std::cout << opts.sep << t;
  }
  for (auto t : competitionTimes) {
    std::cout << opts.sep << t;
  }
  for (auto t : iterationTimes) {
    std::cout << opts.sep << t;
  }

  std::cout << std::endl;

  return 0;
}

void print_header(const RunOptions &opts) {
  fmt::print("benchmark{0}bs{0}graph{0}nodes{0}edges{0}tris", opts.sep);
  for (int i = 0; i < opts.iters; ++i) {
    fmt::print("{}readMostly{}", opts.sep, i);
  }
  for (int i = 0; i < opts.iters; ++i) {
    fmt::print("{}accessedBy{}", opts.sep, i);
  }
  for (int i = 0; i < opts.iters; ++i) {
    fmt::print("{}prefetchAsync{}", opts.sep, i);
  }
  for (int i = 0; i < opts.iters; ++i) {
    fmt::print("{}count{}", opts.sep, i);
  }
  for (int i = 0; i < opts.iters; ++i) {
    fmt::print("{}count_teps{}", opts.sep, i);
  }
  fmt::print("\n");
}

int main(int argc, char **argv) {

  pangolin::init();

  RunOptions opts;
  opts.sep = ",";
  opts.dimBlock = 512;
  opts.iters = 1;
  opts.readMostly = false;
  opts.accessedBy = false;
  opts.prefetchAsync = false;

  bool help = false;
  bool debug = false;
  bool verbose = false;
  bool onlyPrintHeader = false;
  bool wide = false;

  clara::Parser cli;
  cli = cli | clara::Help(help);
  cli = cli | clara::Opt(debug)["--debug"]("print debug messages to stderr");
  cli = cli | clara::Opt(verbose)["--verbose"]("print verbose messages to stderr");
  cli = cli | clara::Opt(onlyPrintHeader)["--header"]("print the header for the times output and quit");
  cli = cli | clara::Opt(wide)["--wide"]("64-bit node IDs");
  cli = cli | clara::Opt(opts.gpus, "dev ids")["-g"]("gpus to use");
  cli = cli | clara::Opt(opts.dimBlock, "block-dim")["--bs"]("Number of threads in a block");
  cli = cli | clara::Opt(opts.readMostly)["--read-mostly"]("mark data as read-mostly by all gpus before kernel");
  cli = cli | clara::Opt(opts.accessedBy)["--accessed-by"]("mark data as accessed-by all GPUs before kernel");
  cli = cli | clara::Opt(opts.prefetchAsync)["--prefetch-async"]("prefetch data to all GPUs before kernel");
  cli = cli | clara::Opt(opts.iters, "N")["-n"]("number of counts");
  cli = cli | clara::Arg(opts.path, "graph file")("Path to adjacency list").required();

  auto result = cli.parse(clara::Args(argc, argv));
  if (!result) {
    LOG(error, "Error in command line: {}", result.errorMessage());
    exit(1);
  }

  if (help) {
    std::cout << cli;
    return 0;
  }

  // set logging level
  if (verbose) {
    pangolin::logger::set_level(pangolin::logger::Level::TRACE);
  } else if (debug) {
    pangolin::logger::set_level(pangolin::logger::Level::DEBUG);
  }

  // log command line before much else happens
  {
    std::string cmd;
    for (int i = 0; i < argc; ++i) {
      if (i != 0) {
        cmd += " ";
      }
      cmd += argv[i];
    }
    LOG(debug, cmd);
  }
  LOG(debug, "pangolin version: {}.{}.{}", PANGOLIN_VERSION_MAJOR, PANGOLIN_VERSION_MINOR, PANGOLIN_VERSION_PATCH);
  LOG(debug, "pangolin branch:  {}", PANGOLIN_GIT_REFSPEC);
  LOG(debug, "pangolin sha:     {}", PANGOLIN_GIT_HASH);
  LOG(debug, "pangolin changes: {}", PANGOLIN_GIT_LOCAL_CHANGES);

#ifndef NDEBUG
  LOG(warn, "Not a release build");
#endif

  if (onlyPrintHeader) {
    print_header(opts);
    return 0;
  }
  if (wide) {
    return run<uint64_t>(opts);
  } else {
    return run<uint32_t>(opts);
  }
}
