#include "hip/hip_runtime.h"
/*!

Count triangles using CUSparse

*/

#include <iostream>
#include <vector>

#include <nvToolsExt.h>

#include <clara/clara.hpp>
#include <fmt/format.h>

#include "pangolin/configure.hpp"
#include "pangolin/file/tsv.hpp"
#include "pangolin/init.hpp"
#include "pangolin/sparse/csr_val.hpp"
#include "pangolin/algorithm/csr/tc_cusparse.hpp"
#include "pangolin/cuda_cxx/stream.hpp"

struct RunOptions {
  int iters;
  std::vector<int> gpus;
  std::string path; //!< path for graph
  std::string sep;  //!< seperator for output

  bool readMostly;
  bool accessedBy;
  bool prefetchAsync;
  bool shrinkToFit;
  bool preCountBarrier;
};

void print_header(const RunOptions &opts) {
  fmt::print("bmark{0}bs{0}gpus{0}graph{0}nodes{0}edges{0}tris", opts.sep);
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}total_time{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}total_teps{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}gpu_time{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}gpu_teps{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}count_time{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}count_teps{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}kernel_time{}", opts.sep, i);
  }
  for (auto i = 0; i < opts.iters; ++i) {
    fmt::print("{}kernel_teps{}", opts.sep, i);
  }
  fmt::print("\n");
}

template <typename V> void print_vec(const V &vec, const std::string &sep) {
  for (const auto &e : vec) {
    fmt::print("{}{}", sep, e);
  }
}

int run(RunOptions &opts) {

  // CUSparse uses integers for indices
  typedef int NodeIndex;
  typedef int EdgeIndex;
  typedef float Val;
  typedef pangolin::WeightedDiEdge<NodeIndex, Val> GraphEdge;
  typedef pangolin::CSR<NodeIndex, EdgeIndex, Val> CSR;
  typedef pangolin::file::TSV TSV;
  typedef TSV::edge_type FileEdge;
  typedef pangolin::CUSparseTC TC;
  using pangolin::Stream;

  auto gpus = opts.gpus;
  if (gpus.empty()) {
    LOG(warn, "no GPUs provided on command line, using GPU 0");
    gpus.push_back(0);
  }

  // create a stream for each GPU
  std::vector<Stream> streams;
  for (const auto &gpu : gpus) {
    streams.push_back(Stream(gpu));
    LOG(debug, "created stream {} for gpu {}", streams.back(), gpu);
  }

  std::vector<double> totalTimes;
  std::vector<double> gpuTimes;
  std::vector<double> countTimes;
  std::vector<double> kernelTimes;
  uint64_t nnz;
  uint64_t numRows;
  uint64_t tris;
  // create csr and count `opts.iters` times
  for (int i = 0; i < opts.iters; ++i) {

    const auto totalStart = std::chrono::system_clock::now();

    // read data
    TSV file(opts.path);
    std::vector<FileEdge> fileEdges = file.read_edges();
    double elapsed = (std::chrono::system_clock::now() - totalStart).count() / 1e9;
    LOG(info, "read_data time {}s", elapsed);
    LOG(debug, "read {} edges", fileEdges.size());

    // build CSR
    CSR csr;
    for (auto fileEdge : fileEdges) {
      GraphEdge graphEdge;
      graphEdge.src = fileEdge.src;
      graphEdge.dst = fileEdge.dst;
      graphEdge.val = fileEdge.val;
      if (graphEdge.src > graphEdge.dst) {
        csr.add_next_edge(graphEdge);
      }
    }
    csr.finish_edges();

    if (opts.shrinkToFit) {
      LOG(debug, "shrink CSR");
      csr.shrink_to_fit();
    }

    elapsed = (std::chrono::system_clock::now() - totalStart).count() / 1e9;
    LOG(info, "io/csr time {}s", elapsed);
    LOG(debug, "CSR nnz = {} rows = {}", csr.nnz(), csr.num_rows());
    LOG(debug, "CSR cap = {}MB size = {}MB", csr.capacity_bytes() / 1024 / 1024, csr.size_bytes() / 1024 / 1024);

    const auto gpuStart = std::chrono::system_clock::now();

    // read-mostly
    nvtxRangePush("read-mostly");
    auto start = std::chrono::system_clock::now();
    if (opts.readMostly) {
      csr.read_mostly();
    }
    elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    nvtxRangePop();
    LOG(info, "read-mostly CSR time {}s", elapsed);

    // accessed-by
    start = std::chrono::system_clock::now();
    if (opts.accessedBy) {
      for (const auto &gpu : gpus) {
        csr.accessed_by(gpu);
      }
    }
    elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    LOG(info, "accessed-by CSR time {}s", elapsed);

    // prefetch
    start = std::chrono::system_clock::now();
    if (opts.prefetchAsync) {
      for (size_t gpuIdx = 0; gpuIdx < gpus.size(); ++gpuIdx) {
        auto &gpu = gpus[gpuIdx];
        hipStream_t stream = streams[gpuIdx].stream();
        csr.prefetch_async(gpu, stream);
      }
    }
    elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    LOG(info, "prefetch CSR time {}s", elapsed);

    if (opts.preCountBarrier) {
      LOG(debug, "sync streams after hints");
      for (auto &stream : streams) {
        stream.sync();
      }
    }

    // count triangles
    nvtxRangePush("count");
    const auto countStart = std::chrono::system_clock::now();
    TC counter(gpus[0]);
    uint64_t count = counter.count_sync(csr);


    const auto stop = std::chrono::system_clock::now();
    nvtxRangePop(); // count

    // record graph stats
    // tris = total;
    nnz = csr.nnz();
    numRows = csr.num_rows();

    const double totalElapsed = (stop - totalStart).count() / 1e9;
    const double gpuElapsed = (stop - gpuStart).count() / 1e9;
    const double countElapsed = (stop - countStart).count() / 1e9;
    LOG(info, "total time {}s ({} teps)", totalElapsed, nnz / totalElapsed);
    LOG(info, "gpu time   {}s ({} teps)", gpuElapsed, nnz / gpuElapsed);
    LOG(info, "count time {}s ({} teps)", countElapsed, nnz / countElapsed);
    totalTimes.push_back(totalElapsed);
    gpuTimes.push_back(gpuElapsed);
    countTimes.push_back(countElapsed);

    // for (auto &counter : counters) {
    //   double secs = counter.kernel_time();
    //   int dev = counter.device();
    //   LOG(info, "gpu {} kernel time {}s ({} teps)", dev, secs, nnz / secs);
    // }
    // if (counters.size() == 1) {
    //   kernelTimes.push_back(counters[0].kernel_time());
    // } else {
    //   kernelTimes.push_back(0);
    // }
  }

  if (opts.iters > 0) {
    fmt::print("binary");
    std::string gpuStr;
    for (auto gpu : gpus) {
      gpuStr += std::to_string(gpu);
    }
    fmt::print("{}{}", opts.sep, gpuStr);
    fmt::print("{}{}", opts.sep, opts.path);
    fmt::print("{}{}", opts.sep, numRows);
    fmt::print("{}{}", opts.sep, nnz);
    fmt::print("{}{}", opts.sep, tris);

    print_vec(totalTimes, opts.sep);
    for (const auto &s : totalTimes) {
      fmt::print("{}{}", opts.sep, nnz / s);
    }
    print_vec(gpuTimes, opts.sep);
    for (const auto &s : gpuTimes) {
      fmt::print("{}{}", opts.sep, nnz / s);
    }
    print_vec(countTimes, opts.sep);
    for (const auto &s : countTimes) {
      fmt::print("{}{}", opts.sep, nnz / s);
    }
    print_vec(kernelTimes, opts.sep);
    for (const auto &s : kernelTimes) {
      fmt::print("{}{}", opts.sep, nnz / s);
    }

    fmt::print("\n");
  }

  return 0;
}

int main(int argc, char **argv) {

  pangolin::init();

  RunOptions opts;
  opts.sep = ",";
  opts.iters = 1;
  opts.shrinkToFit = false;
  opts.readMostly = false;
  opts.accessedBy = false;
  opts.prefetchAsync = false;
  opts.preCountBarrier = true;

  bool help = false;
  bool debug = false;
  bool verbose = false;
  bool onlyPrintHeader = false;

  clara::Parser cli;
  cli = cli | clara::Help(help);
  cli = cli | clara::Opt(debug)["--debug"]("print debug messages to stderr");
  cli = cli | clara::Opt(verbose)["--verbose"]("print verbose messages to stderr");
  cli = cli | clara::Opt(onlyPrintHeader)["--header"]("print the header for the times output and quit");
  cli = cli | clara::Opt(opts.gpus, "dev ids")["-g"]("gpus to use");
  cli = cli | clara::Opt(opts.shrinkToFit)["--shrink-to-fit"]("shrink allocations to fit data");
  cli = cli | clara::Opt(opts.readMostly)["--read-mostly"]("mark data as read-mostly by all gpus before kernel");
  cli = cli | clara::Opt(opts.accessedBy)["--accessed-by"]("mark data as accessed-by all GPUs before kernel");
  cli = cli | clara::Opt(opts.prefetchAsync)["--prefetch-async"]("prefetch data to all GPUs before kernel");
  cli = cli | clara::Opt(opts.iters, "N")["-n"]("number of counts");
  cli = cli | clara::Arg(opts.path, "graph file")("Path to adjacency list").required();

  auto result = cli.parse(clara::Args(argc, argv));
  if (!result) {
    LOG(error, "Error in command line: {}", result.errorMessage());
    exit(1);
  }

  if (help) {
    std::cout << cli;
    return 0;
  }

  // set logging level
  if (verbose) {
    pangolin::logger::set_level(pangolin::logger::Level::TRACE);
  } else if (debug) {
    pangolin::logger::set_level(pangolin::logger::Level::DEBUG);
  }

  // log command line before much else happens
  {
    std::string cmd;
    for (int i = 0; i < argc; ++i) {
      if (i != 0) {
        cmd += " ";
      }
      cmd += argv[i];
    }
    LOG(debug, cmd);
  }
  LOG(debug, "pangolin version: {}.{}.{}", PANGOLIN_VERSION_MAJOR, PANGOLIN_VERSION_MINOR, PANGOLIN_VERSION_PATCH);
  LOG(debug, "pangolin branch:  {}", PANGOLIN_GIT_REFSPEC);
  LOG(debug, "pangolin sha:     {}", PANGOLIN_GIT_HASH);
  LOG(debug, "pangolin changes: {}", PANGOLIN_GIT_LOCAL_CHANGES);

#ifndef NDEBUG
  LOG(warn, "Not a release build");
#endif

  if (onlyPrintHeader) {
    print_header(opts);
    return 0;
  }
  return run(opts);
}
