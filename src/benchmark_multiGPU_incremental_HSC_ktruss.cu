#include "hip/hip_runtime.h"
#include <mpi.h>

#include <fmt/format.h>
#include <iostream>

#include "omp.h"

#include<map>

#include <nvToolsExt.h>

#include "clara/clara.hpp"
#include "pangolin/pangolin.cuh"
#include "pangolin/pangolin.hpp"
#include "pangolin/algorithm/zero.cuh"

#define UT uint32_t

int getMaxK(std::map<UT, int> degree)
{
	typedef std::map<UT, int>::reverse_iterator  it_type;
	int maxK = 0;
	int reverseCount = 0;
	bool getNext = false;
	for (it_type m = degree.rbegin(); m != degree.rend(); m++)
	{
		int degree = m->first;
		int proposedKmax = degree + 1;

		reverseCount += m->second;

		if (reverseCount >= proposedKmax)
		{
			maxK = proposedKmax;
			break;
		}
	}

	return maxK;
}


int main(int argc, char **argv) {

  pangolin::init(); 
  pangolin::Config config;

  std::vector<int> gpus;
  std::string path;
  int iters = 1;
  bool help = false;
  bool debug = false;
  bool verbose = false;

  bool readMostly = false;
  bool accessedBy = false;
  bool prefetchAsync = false;

  clara::Parser cli;
  cli = cli | clara::Help(help);
  cli = cli | clara::Opt(debug)["--debug"]("print debug messages to stderr");
  cli = cli |
        clara::Opt(verbose)["--verbose"]("print verbose messages to stderr");
  cli = cli | clara::Opt(gpus, "ids")["-g"]("gpus to use");
  cli = cli | clara::Opt(readMostly)["--read-mostly"](
                  "mark data as read-mostly by all gpus before kernel");
  cli = cli | clara::Opt(accessedBy)["--accessed-by"](
                  "mark data as accessed-by all GPUs before kernel");
  cli = cli | clara::Opt(prefetchAsync)["--prefetch-async"](
                  "prefetch data to all GPUs before kernel");
  cli = cli | clara::Opt(iters, "N")["-n"]("number of counts");
  cli =
      cli | clara::Arg(path, "graph file")("Path to adjacency list").required();

  auto result = cli.parse(clara::Args(argc, argv));
  if (!result) {
    LOG(error, "Error in command line: {}", result.errorMessage());
    exit(1);
  }

  if (help) {
    std::cout << cli;
    return 0;
  }

  // set logging level
  if (verbose) {
    pangolin::logger::set_level(pangolin::logger::Level::TRACE);
  } else if (debug) {
    pangolin::logger::set_level(pangolin::logger::Level::DEBUG);
  }

  // log command line before much else happens
  {
    std::string cmd;
    for (int i = 0; i < argc; ++i) {
      if (i != 0) {
        cmd += " ";
      }
      cmd += argv[i];
    }
    LOG(debug, cmd);
  }
  LOG(debug, "pangolin version: {}.{}.{}", PANGOLIN_VERSION_MAJOR,
      PANGOLIN_VERSION_MINOR, PANGOLIN_VERSION_PATCH);
  LOG(debug, "pangolin branch:  {}", PANGOLIN_GIT_REFSPEC);
  LOG(debug, "pangolin sha:     {}", PANGOLIN_GIT_HASH);
  LOG(debug, "pangolin changes: {}", PANGOLIN_GIT_LOCAL_CHANGES);

#ifndef NDEBUG
  LOG(warn, "Not a release build");
#endif

  if (gpus.empty()) {
    LOG(warn, "no GPUs provided on command line, using GPU 0");
    gpus.push_back(0);
  }

  // read data
  auto start = std::chrono::system_clock::now();
  pangolin::EdgeListFile file(path);

  std::vector<pangolin::EdgeTy<UT>> edges;
  std::vector<pangolin::EdgeTy<UT>> fileEdges;
  while (file.get_edges(fileEdges, 10)) {
    edges.insert(edges.end(), fileEdges.begin(), fileEdges.end());
  }
  double elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
  LOG(info, "read_data time {}s", elapsed);
  LOG(debug, "read {} edges", edges.size());

  // create csr and count `iters` times
  std::vector<double> times;
  uint64_t nnz;

  // create csr
  start = std::chrono::system_clock::now();
  auto upperTriangular = [](pangolin::EdgeTy<UT> e) {
    return true; //e.first < e.second;
  };
  auto csr = pangolin::COO<UT>::from_edges(edges.begin(), edges.end(),
                                                  upperTriangular);
  LOG(debug, "nnz = {}", csr.nnz());
  elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
  LOG(info, "create CSR time {}s", elapsed);

  
  // accessed-by
  nvtxRangePush("accessed-by");
  start = std::chrono::system_clock::now();
  if (accessedBy) {
    for (const auto &gpu : gpus) {
      csr.accessed_by(gpu);
      CUDA_RUNTIME(hipSetDevice(gpu));
      CUDA_RUNTIME(hipDeviceSynchronize());
    }
  }
  elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
  nvtxRangePop();
  LOG(info, "accessed-by CSR time {}s", elapsed);

  // prefetch
  nvtxRangePush("prefetch");
  start = std::chrono::system_clock::now();
  if (prefetchAsync) {
    for (const auto &gpu : gpus) {
      csr.prefetch_async(gpu);
      CUDA_RUNTIME(hipSetDevice(gpu));
      CUDA_RUNTIME(hipDeviceSynchronize());
    }
  }
  elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
  nvtxRangePop();
  LOG(info, "prefetch CSR time {}s", elapsed);

  // count triangles
  nvtxRangePush("count");
  //start = std::chrono::system_clock::now();

  {
    start = std::chrono::system_clock::now();

    //csr.read_mostly();
    UT *rowPtr = csr.rowPtr_.data();
    UT *rowInd = csr.rowInd_.data();
    UT *colInd = csr.colInd_.data();

    int numEdges = csr.nnz();
    int numGpus = gpus.size();
    int numNodes = csr.num_nodes();

    int edgesPerGPU = (numEdges + numGpus - 1) / numGpus;
    pangolin::Vector<UT> uSrcKp(numEdges);
    pangolin::Vector<UT> uDstKp(numEdges);
    pangolin::Vector<UT> uReversed(numEdges);
    
    printf("NNZ=%d\n", numEdges);

    // create async counters
    std::vector<pangolin::MultiGPU_Ktruss_Incremental> trussCounters;
    for (int dev : gpus) {
      LOG(info, "create device {} counter", dev);
      auto counter = pangolin::MultiGPU_Ktruss_Incremental(numEdges, dev);
      counter.CreateWorkspace(numEdges);
      trussCounters.push_back(counter);
      counter.InitializeWorkSpace_async(numEdges);
    }

    int edgeStart = 0;
    for (auto &counter : trussCounters) 
    { 

      counter.selectedOut[0] = numEdges;

      const size_t edgeStop = std::min(edgeStart + edgesPerGPU, numEdges);
      const size_t edgesToProcess = edgeStop - edgeStart;
      counter.Inialize_Unified_async(edgeStart, edgesToProcess, rowPtr, rowInd, colInd, uSrcKp.data(), uReversed.data());
      edgeStart += edgesPerGPU;
    }

    UT *ptrSrc, *ptrDst;
    UT *s1, *d1, *s2, *d2;

    s1 = rowInd;
		d1 = colInd;

		s2 = uSrcKp.data();
		d2 = uDstKp.data();

		ptrSrc = s1;
		ptrDst = d1;
    
    int kmin = 3;
    int kmax=-1;
    constexpr int dimBlock = 32; //For edges and nodes
    int dimGridEdges = (numEdges + dimBlock - 1) / dimBlock;

    if(numGpus > 1)
    {
      CUDA_RUNTIME(hipMemAdvise(ptrSrc, numEdges * sizeof(UT), hipMemAdviseSetReadMostly, 0 /* ignored */));
      CUDA_RUNTIME(hipMemAdvise(ptrDst, numEdges * sizeof(UT), hipMemAdviseSetReadMostly, 0 /* ignored */));
      CUDA_RUNTIME(hipMemAdvise(rowPtr, (numNodes+1) * sizeof(UT), hipMemAdviseSetReadMostly, 0 /* ignored */));
      CUDA_RUNTIME(hipMemAdvise(uReversed.data(), numEdges * sizeof(UT), hipMemAdviseSetReadMostly, 0 /* ignored */));
    }

    while(true)
    {

      LOG(info, "kmin={}, remaining edges={}", kmin, numEdges);

      bool firstTry = true;
      for (auto &counter : trussCounters) 
      {
        counter.setDevice();
        counter.hnumaffected[0] = 1;
        CUDA_RUNTIME(hipMemsetAsync(counter.gnumaffected,0,sizeof(UT),counter.stream()));
      }

      bool assumpAffected = true;
      
      /*nvtxRangePush("kernel per k");
      start = std::chrono::system_clock::now();*/
      while(assumpAffected)
      {
        assumpAffected = false;
        for (int i=0; i<numGpus;i++) 
        { 
          auto& counter = trussCounters[i];
          counter.setDevice();
          if(counter.hnumaffected[0]>0)
          {
              core_direct<dimBlock><<<dimGridEdges,dimBlock,0,counter.stream()>>>(counter.gnumdeleted, 
                counter.gnumaffected, kmin+i, 0, numEdges,
                rowPtr, ptrSrc, ptrDst, counter.gKeep, counter.gAffected, uReversed.data(), firstTry, 1);
  
            //Copy to host
            CUDA_RUNTIME(hipMemcpyAsync(counter.hnumaffected, counter.gnumaffected, sizeof(UT), hipMemcpyDeviceToHost, counter.stream()));
            CUDA_RUNTIME(hipMemcpyAsync(counter.hnumdeleted, counter.gnumdeleted, sizeof(UT), hipMemcpyDeviceToHost, counter.stream()));

            //Set gpu data to zeros
            CUDA_RUNTIME(hipMemsetAsync(counter.gnumdeleted,0,sizeof(UT),counter.stream()));
            CUDA_RUNTIME(hipMemsetAsync(counter.gnumaffected,0,sizeof(UT),counter.stream()));
          }
        }
       
        for (int i=0; i<numGpus;i++) 
        { 
          auto& counter = trussCounters[i];
          counter.setDevice();
          counter.sync();
          assumpAffected = assumpAffected || (counter.hnumaffected[0]>0);
          counter.percentage_deleted_k = (counter.hnumdeleted[0])*1.0/numEdges;
        }
        firstTry = false;
      }
  

      bool foundKmax = false;
      int fallBackGPU = -1;
      for (int i=0; i<numGpus;i++) 
      { 
        auto& counter = trussCounters[i];
        counter.setDevice();

        if(numEdges - counter.hnumdeleted[0] > 0)
        {
          kmax = kmin + i;
          fallBackGPU = i;

        }
        else
        {
          foundKmax = true;
          break;
        }
      }

      

      kmin += numGpus;
      int counter = 0; 
     
      if(!foundKmax)
      {
        auto& c = trussCounters[fallBackGPU];
        float percDeleted = (c.hnumdeleted[0])*1.0/numEdges;
        if(c.hnumdeleted[0] > 1000)
        {
          if(numGpus > 1)
          {
            CUDA_RUNTIME(hipMemAdvise(rowPtr, (numNodes+1) * sizeof(UT), hipMemAdviseUnsetReadMostly, 0));
            CUDA_RUNTIME(hipMemAdvise(uReversed.data(), numEdges * sizeof(UT), hipMemAdviseUnsetReadMostly, 0));
          }


          //each gpu stores latest keep
          
          c.setDevice();
          void     *d_temp_storage = NULL;
          size_t   temp_storage_bytes = 0;
          
          hipcub::DevicePartition::Flagged(d_temp_storage, temp_storage_bytes, s1, c.gKeep, s2, c.selectedOut, numEdges, c.stream());
          CUDA_RUNTIME(hipMalloc(&d_temp_storage, temp_storage_bytes));
          hipcub::DevicePartition::Flagged(d_temp_storage, temp_storage_bytes, s1, c.gKeep, s2, c.selectedOut, numEdges, c.stream());
          hipcub::DevicePartition::Flagged(d_temp_storage, temp_storage_bytes, d1, c.gKeep, d2, c.selectedOut, numEdges, c.stream());
          CUDA_RUNTIME(hipFree(d_temp_storage));

          hipDeviceSynchronize();
          CUDA_RUNTIME(hipGetLastError());

          if(numGpus > 1)
          {
            CUDA_RUNTIME(hipMemAdvise(s1, numEdges * sizeof(UT), hipMemAdviseUnsetReadMostly, 0));
            CUDA_RUNTIME(hipMemAdvise(d1, numEdges * sizeof(UT), hipMemAdviseUnsetReadMostly, 0));
          
            CUDA_RUNTIME(hipMemAdvise(s2, numEdges * sizeof(UT), hipMemAdviseSetReadMostly, 0 ));
            CUDA_RUNTIME(hipMemAdvise(d2, numEdges * sizeof(UT), hipMemAdviseSetReadMostly, 0 ));
          }

          numEdges = c.selectedOut[0];
          edgesPerGPU = (numEdges + numGpus - 1) / numGpus;
          dimGridEdges =  (numEdges + dimBlock - 1) / dimBlock;

          ptrSrc = s2;
          s2 = s1;
          s1 = ptrSrc;

          ptrDst = d2;
          d2 = d1;
          d1 = ptrDst;
          
          c.setDevice();
          RebuildArrays<dimBlock><<<dimGridEdges,dimBlock,0,c.stream()>>>(0, numEdges, numEdges, rowPtr, ptrSrc); 
          RebuildReverse<dimBlock><<<dimGridEdges,dimBlock,0,c.stream()>>>(0, numEdges, rowPtr, ptrSrc, ptrDst, uReversed.data());
          for (auto &counter : trussCounters)
          { 
            counter.setDevice();
            counter.InitializeWorkSpace_async(numEdges);
          } 

          
          if(numGpus > 1)
          {
            CUDA_RUNTIME(hipMemAdvise(rowPtr, (numNodes+1) * sizeof(UT), hipMemAdviseSetReadMostly, 0 ));
            CUDA_RUNTIME(hipMemAdvise(uReversed.data(), numEdges * sizeof(UT), hipMemAdviseSetReadMostly, 0 ));
          }

          CUDA_RUNTIME(hipGetLastError());
          

          for (auto &counter : trussCounters)
          { 
            counter.sync();
            
          } 
        }
      }
      else{
        break;
      }
    } 

    //printf("New Kmin = %d, New Kmax=%d\n", newKmin, newKmax);
    for (auto &counter : trussCounters)
      counter.free();

    elapsed = (std::chrono::system_clock::now() - start).count() / 1e9;
    nvtxRangePop();
    LOG(info, "count time {}s", elapsed);
    LOG(info, "MOHA {} ktruss ({} teps)", kmax, csr.nnz() / elapsed);
    times.push_back(elapsed);
    //tris = total;
    nnz = csr.nnz();

    //std::cout << path << ",\t" << nnz << ",\t" << tris;
    for (const auto &t : times) {
      std::cout << ",\t" << t;
    }
    std::cout << std::endl;
  }

  return 0;
}
